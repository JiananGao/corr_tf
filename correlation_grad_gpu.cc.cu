
#include <hip/hip_runtime.h>
/* Copyright 2015 The TensorFlow Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "correlation_param.h"


// Device functions only run on GPU and are typically inlined
__device__
int getThreadIdx_3D_3D(){
    int threadId = (threadIdx.z * (blockDim.x * blockDim.y))
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void CorrelationGradKernel(const float* a, const float*b,const float*grad,float* out_a,float*out_b, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* g_offset_list)  {
    int one_d_size   = depth;
    int two_d_size   = one_d_size*num_cols;
    int three_d_size = two_d_size*num_rows;

    int out1 = num_offsets;
    int out2 = num_cols * out1;
    int out3 = num_rows * out2;

    int num_offset_ints = 2*num_offsets;
    // Copy the offset list into shared memory to speed up access
    __shared__ int offset_list[CORRELATION_OPERATOR_LIST_SIZE];
    int mem_index = getThreadIdx_3D_3D();
    int total_block_size = blockDim.x * blockDim.y * blockDim.z;
    for( ; mem_index < num_offset_ints; mem_index+= total_block_size)
    {
       offset_list[mem_index] = g_offset_list[mem_index];
    }
    __syncthreads();

    for (int i = blockIdx.z * blockDim.z + threadIdx.z; i < batch_size; i+= blockDim.z * gridDim.z) {
        int b_root = i*three_d_size;
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < num_rows; j += blockDim.x * gridDim.x) {
          for (int k = blockIdx.y*blockDim.y + threadIdx.y; k < num_cols; k += blockDim.y * gridDim.y) {
            int grad_root = out3*i + out2*j+out1*k;
            int a_root = three_d_size*i + two_d_size*j+one_d_size * k;
            for( int m = 0 ; m < depth; m++) {
              int a_index = a_root+m;
              for (int l =0; l < num_offsets; l++ ) {
                int j_offset = offset_list[2*l];
                int k_offset = offset_list[2*l+1];
                int min_j = 0;
                int max_j = num_rows;
                int min_k = 0;
                int max_k = num_cols;
                if(j_offset < 0){
                    min_j = -1*j_offset;
                }else{
                    max_j -= j_offset;
                }
                if(k_offset < 0){
                    min_k = -1*k_offset;
                }else{
                    max_k -= k_offset;
                }

                int grad_index =  grad_root+ l;
                if( j >= min_j && j < max_j  && k >= min_k && k < max_k)
                {
                    int b_j = j+j_offset;
                    int b_k = k+k_offset;
                    int b_index = b_root + two_d_size*b_j + one_d_size * b_k +m;
                    float current_coefficient = grad[grad_index]/ depth;
                    out_a[a_index]+= current_coefficient*b[b_index];
                    // THIS will clobber out_b occasionally, as different threads will try to write to the same b_index
                    out_b[b_index]+= current_coefficient*a[a_index];
	    
                }
              }
             }
           }
        }
      }

}

__global__ void CorrelationGradAKernel(const float* a, const float*b,const float*grad,float* out_a,float*out_b, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* g_offset_list)  {
    int one_d_size   = depth;
    int two_d_size   = one_d_size*num_cols;
    int three_d_size = two_d_size*num_rows;

    int out1 = num_offsets;
    int out2 = num_cols * out1;
    int out3 = num_rows * out2;

    int num_offset_ints = 2*num_offsets;
    // Copy the offset list into shared memory to speed up access
    __shared__ int offset_list[CORRELATION_OPERATOR_LIST_SIZE];
    int mem_index = getThreadIdx_3D_3D();
    int total_block_size = blockDim.x * blockDim.y * blockDim.z;
    for( ; mem_index < num_offset_ints; mem_index+= total_block_size)
    {
       offset_list[mem_index] = g_offset_list[mem_index];
    }
    
    __syncthreads();

    for (int i = blockIdx.z * blockDim.z + threadIdx.z; i < batch_size; i+= blockDim.z * gridDim.z) {
        int b_root = i*three_d_size;
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < num_rows; j += blockDim.x * gridDim.x) {
          for (int k = blockIdx.y*blockDim.y + threadIdx.y; k < num_cols; k += blockDim.y * gridDim.y) {
            int grad_root = out3*i + out2*j+out1*k;
            int a_root = three_d_size*i + two_d_size*j+one_d_size * k;
            for( int m = 0 ; m < depth; m++) {
              int a_index = a_root+m;
              for (int l =0; l < num_offsets; l++ ) {
                int j_offset = offset_list[2*l];
                int k_offset = offset_list[2*l+1];
                int min_j = 0;
                int max_j = num_rows;
                int min_k = 0;
                int max_k = num_cols;
                if(j_offset < 0){
                    min_j = -1*j_offset;
                }else{
                    max_j -= j_offset;
                }
                if(k_offset < 0){
                    min_k = -1*k_offset;
                }else{
                    max_k -= k_offset;
                }

                int grad_index =  grad_root+ l;
                if( j >= min_j && j < max_j  && k >= min_k && k < max_k)
                {
                    int b_j = j+j_offset;
                    int b_k = k+k_offset;
                    int b_index = b_root + two_d_size*b_j + one_d_size * b_k +m;
                    float current_coefficient = grad[grad_index]/ depth;
                    out_a[a_index]+= current_coefficient*b[b_index];
                }
              }
             }
           }
        }
      }

}

__global__ void CorrelationGradBKernel(const float* a, const float*b,const float*grad,float* out_a,float*out_b, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* g_offset_list)  {
    int one_d_size   = depth;
    int two_d_size   = one_d_size*num_cols;
    int three_d_size = two_d_size*num_rows;

    int out1 = num_offsets;
    int out2 = num_cols * out1;
    int out3 = num_rows * out2;

    int num_offset_ints = 2*num_offsets;
    // Copy the offset list into shared memory to speed up access
    __shared__ int offset_list[CORRELATION_OPERATOR_LIST_SIZE];
    int mem_index = getThreadIdx_3D_3D();
    int total_block_size = blockDim.x * blockDim.y * blockDim.z;
    for( ; mem_index < num_offset_ints; mem_index+= total_block_size)
    {
       offset_list[mem_index] = g_offset_list[mem_index];
    }
    
    __syncthreads();

    for (int i = blockIdx.z * blockDim.z + threadIdx.z; i < batch_size; i+= blockDim.z * gridDim.z) {
        for (int b_j = blockIdx.x * blockDim.x + threadIdx.x; b_j < num_rows; b_j += blockDim.x * gridDim.x) {
          for (int b_k = blockIdx.y*blockDim.y + threadIdx.y; b_k < num_cols; b_k += blockDim.y * gridDim.y) {
            int b_root = i*three_d_size + two_d_size*b_j + one_d_size * b_k;

              for (int l =0; l < num_offsets; l++ ) {
                int j_offset = offset_list[2*l];
                int k_offset = offset_list[2*l+1];
                int j = b_j - j_offset;
                int k = b_k - k_offset;
                int min_j = 0;
                int max_j = num_rows;
                int min_k = 0;
                int max_k = num_cols;
                if(j_offset < 0){
                    min_j = -1*j_offset;
                }else{
                    max_j -= j_offset;
                }
                if(k_offset < 0){
                    min_k = -1*k_offset;
                }else{
                    max_k -= k_offset;
                }
                if( j >= min_j && j < max_j  && k >= min_k && k < max_k)
                {
                    int grad_root = out3*i + out2*j+out1*k;
                    int grad_index =  grad_root+ l;
                    int a_root = three_d_size*i + two_d_size*j+one_d_size * k;
                    for( int m = 0 ; m < depth; m++) {
                      int b_index = b_root +m;
                      int a_index = a_root+m;
                      float current_coefficient = grad[grad_index]/ depth;
                      out_b[b_index]+= current_coefficient*a[a_index];
	    
                    }
              }
             }
           }
        }
      }

}


/// Take the tensor arrays (which are allocated on the GPU by TensorFlow's  context->allocate_output() call )
/// and spawn the correct number of CUDA threads on the GPU
void CorrelationGradKernelLauncher(const float* a, const float*b, const float*grad, float* out_a,float*out_b, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* offset_list) {
  // Move the offset array to GPU, since this one was allocated by the std::vector on the CPU side 
  int *offset_array;
  cudaMalloc(&offset_array, 2*num_offsets * sizeof(int)); 
  cudaMemcpy(offset_array, offset_list, 2*num_offsets*sizeof(int), cudaMemcpyHostToDevice);

  // Zero out the outputs, which we assume were allocated on the GPU by the context->allocate_output() call
  size_t out_size = batch_size*num_rows*num_cols*depth*sizeof(float);
  cudaMemset(out_a,0,out_size);
  cudaMemset(out_b,0,out_size);

  // Address the image in blocks of size  1 (batch)x 16 (height)x 16 (width) x  num_channel (depth) 
  int mx = 16;
  int my = 16;
  int mz = 1;
  // Calculate how many blocks are needed to cover the whole image. 
  // This math is long-hand for int nz = ceil(batch_size/mz);
  int nz = (batch_size + mz -1)/mz;
  int ny = (num_cols + my - 1)/my;
  int nx = (num_rows + mx -1)/mx;

  // Use CUDA's dim3 structs to contain the block counts and block shapes
  dim3 blocks(nx,ny,nz);
  dim3 threadsPerBlock(mx,my,mz);
  // Call the CUDA Kernel
  // Calculate gradient A and gradient B separately to avoid collisions 
  CorrelationGradAKernel<<<blocks, threadsPerBlock>>>(a, b, grad, out_a,out_b,batch_size,num_rows,num_cols,depth,num_offsets,offset_array);
  CorrelationGradBKernel<<<blocks, threadsPerBlock>>>(a, b, grad, out_a,out_b,batch_size,num_rows,num_cols,depth,num_offsets,offset_array);
}

#endif
